#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16
__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 


/*Function representing the 2nd order differential equation (modify this according to your equation)
  My Equation is:
  */
__device__ double f(double t, double y, double z) {
// Example: dz/dt = d^2y/dt^2 =  = -A*Sin(2y) +B*Sin(wt)*Sin(y) -Cz + D
double A=1, B=2, C=3, D=4, w=5;
return -A*sin(2*y) + B*sin(w*t)*sin(y) -C*z + D;
}

// Euler method implementation
__global__ void eulerMethod(double t0, double y0, double z0, double dt, double t_end) {
double t = t0;
double y = y0;
double z = z0;

while (t <= t_end) {
printf("t = %lf, y = %lf\n", t, y);

double y_next = y + dt * z;
double z_next = z + dt * f(t, y, z);

t += dt;
y = y_next;
z = z_next;
}
}


int main(int argc, char const *argv[])
{
    /* Fixed seed for illustration */
    srand(3333);
    int m=16, n=16, k=16;
   
    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k);
    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);

    // random initialize matrix A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
    }

    // Allocate memory space on the device 
    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
   
    // Launch kernel 
    if(m == n && n == k)
    {
        gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);    
    }
    else
    {
        gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);    
    }
    // Transefr results from device to host 
    hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    // validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            printf("[%d][%d]:%d == [%d][%d]:%d, ", i, j, h_cc[i*k + j], i, j, h_c[i*k + j]);
            if(h_cc[i*k + j] != h_c[i*k + j])
            {
                all_ok = 0;
            }
        }
        //printf("\n");
    }

    
    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}
